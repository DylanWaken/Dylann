//
// Created by Dylan on 9/6/2022.
//

#include "DylannContext.cuh"
#include "tensor/cuTensor.cuh"

namespace dylann {
    
    map<TENSOR_PTR ,cuTensorBase*> tensorsCTX;
    map<TENSOR_PTR ,cuTensorBase*> paramsCTX;   //optimizers will be applied on these
    
    vector<Operation*> forwardOpsCTX;
    vector<Operation*> backwardOpsCTX;
    
    bool regisModeCTX;
    unsigned int tensorIDSeqCTX = 0;
    bool engineAliveCTX = false;
    
    //register mode is automatically started after initialized engine context
    void initEngineContext(){
        *tensorIDSeqG = tensorIDSeqCTX;
        
        cuTensorBase::tensorPoolG = &tensorsCTX;
        cuTensor::instructions = &forwardOpsCTX;
        
        engineAliveCTX = true;
        regisModeCTX = true;
        onModelRegisterG = true;
    }
    
    void beganModelRegister(){
        regisModeCTX = true;
        onModelRegisterG = true;
    }
    
    void allocModelParams(){
        for(auto it : tensorsCTX){
            if(it.second->desc.isParam){
                paramsCTX.insert(it);
            }
        }
    }
    
    void endModelRegister(){
        regisModeCTX = false;
        onModelRegisterG = false;
    }
    
    Sequence* ctx2seq(){
        Sequence* seq;
        hipHostMalloc(&seq, sizeof(Sequence));
        seq->tensorsSeq = tensorsCTX;
        seq->paramsSeq = paramsCTX;
        seq->forwardOpSeq = forwardOpsCTX;
        seq->backwardOpSeq = backwardOpsCTX;
    
        for (auto& op : seq->forwardOpSeq) {
            op->bind(&seq->tensorsSeq);
        }
        
        for (auto& op : seq->backwardOpSeq) {
            op->bind(&seq->tensorsSeq);
        }
    
        tensorsCTX.clear();
        paramsCTX.clear();
        forwardOpsCTX.clear();
        backwardOpsCTX.clear();
    
        *tensorIDSeqG = 0;
        return seq;
    }
    
    Sequence* ctx2SeqExport(){
        Sequence* seq;
        hipHostMalloc(&seq, sizeof(Sequence));
        seq->tensorsSeq = tensorsCTX;
        seq->paramsSeq = paramsCTX;
        seq->forwardOpSeq = forwardOpsCTX;
        seq->backwardOpSeq = backwardOpsCTX;
    
        for (auto& op : seq->forwardOpSeq) {
            op->bind(&seq->tensorsSeq);
        }
    
        for (auto& op : seq->backwardOpSeq) {
            op->bind(&seq->tensorsSeq);
        }
        
        return seq;
    }
} // dylann