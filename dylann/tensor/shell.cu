#include "hip/hip_runtime.h"
//
// Created by Dylan on 8/8/2022.
//

#include "shell.cuh"

namespace dylann{
    cuTensor add(cuTensor& A, cuTensor& B, float alpha, float beta){
        add(A.impl, B.impl, alpha, beta);
        
        GradTracker* t1 = new GRAD_ADD_A(alpha);
        A.gradStack.emplace(&A, t1);
        
        GradTracker* t2 = new GRAD_ADD_B(beta, B.impl);
        A.gradStack.emplace(&B, t2);
        
        return A;
    }
    
    cuTensor scale(cuTensor& A, float alpha){
        scale(A.impl, alpha);
        
        GradTracker* t = new GRAD_SCALE(alpha);
        A.gradStack.emplace(&A, t);
    
        return A;
    }
    
    cuTensor linear(cuTensor& W, cuTensor& B, cuTensor& X, cuTensor& Y){
        linearOp(W.impl, B.impl, X.impl, Y.impl);
        
        GradTracker* t1 = new GRAD_LINEAR(W.impl, B.impl, X.impl);
        Y.gradStack.emplace(&X,t1);
        
        //give Y the access to push grad backward into X
        X.impl->desc.gradSrcUuid = Y.desc().uuid;
        
        return Y;
    }
    
    cuTensor conv2D(cuTensor& X, cuTensor& W, cuTensor& B, cuTensor& Y,
                    int padH, int padW, int strideH, int strideW, int dilationH, int dilationW){
        conv2dOp(X.impl, W.impl, B.impl, Y.impl, padH, padW, strideH, strideW, dilationH, dilationW);
        
        GradTracker* t1 = new GRAD_CONV2D(X.impl, W.impl, B.impl, padH, padW, strideH, strideW, dilationH, dilationW);
        Y.gradStack.emplace(&X,t1);
        
        //give Y the access to push grad backward into X
        X.impl->desc.gradSrcUuid = Y.desc().uuid;
        
        return Y;
    }
    
    //--------------------------------------------------------------------------------
    //Activations
    
    cuTensor relu(cuTensor& X){
        reluOp(X.impl);
        
        GradTracker* t = new GRAD_RELU(X.impl);
        X.gradStack.emplace(&X,t);
        
        return X;
    }
    
    cuTensor relu(cuTensor& X, cuTensor& Y){
        reluOp(X.impl, Y.impl);
        
        GradTracker* t = new GRAD_RELU(X.impl);
        Y.gradStack.emplace(&X,t);
        
        X.impl->desc.gradSrcUuid = Y.desc().uuid;
        
        return Y;
    }
    
    cuTensor randUniform(cuTensor& A, double min, double max){
        return A.randUniform(min, max);
    }
    
    cuTensor randNormal(cuTensor& A, double mean, double stddev){
        return A.randNormal(mean, stddev);
    }
}
