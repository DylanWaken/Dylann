//
// Created by Dylan on 9/4/2022.
//

#include "cuBatchnorm.cuh"

namespace dylann {
    cuTensorBase* batchnormOp(cuTensorBase* X, cuTensorBase* Y, cuTensorBase* runningMean, cuTensorBase* runningVar,
                              cuTensorBase* gamma, cuTensorBase* beta, float eps, float expAvgFactor) {
        assertAllocated({X, Y, runningMean, runningVar, gamma, beta});
        assertOnSameDev({X, Y, runningMean, runningVar, gamma, beta});
        
        hipSetDevice(X->data->deviceID);
        
        float a = 1.0f, b = 0.0f;
        
        //in this function, to save space, we use the grads for runningMean and runningVar as temp storage
        checkCUDNN(hipdnnBatchNormalizationForwardTraining(
                cudnnHdlG,
                HIPDNN_BATCHNORM_SPATIAL,
                &a,
                &b,
                X->desc.cudnnDesc,
                X->data->data,
                Y->desc.cudnnDesc,
                Y->data->data,
                gamma->desc.cudnnDesc,
                gamma->data->data,
                beta->data->data,
                expAvgFactor,
                runningMean->data->data,
                runningVar->data->data,
                eps,
                runningMean->grad->data,
                runningVar->grad->data
                ))
                
        return Y;
    }
    
    cuTensorBase* batchnormInferOp(cuTensorBase* X, cuTensorBase* Y, cuTensorBase* runningMean, cuTensorBase* runningVar,
                                   cuTensorBase* gamma, cuTensorBase* beta, float eps) {
        assertAllocated({X, Y, runningMean, runningVar, gamma, beta});
        assertOnSameDev({X, Y, runningMean, runningVar, gamma, beta});
        
        hipSetDevice(X->data->deviceID);
        
        float a = 1.0f, b = 0.0f;
        
        checkCUDNN(hipdnnBatchNormalizationForwardInference(
                cudnnHdlG,
                HIPDNN_BATCHNORM_SPATIAL,
                &a,
                &b,
                X->desc.cudnnDesc,
                X->data->data,
                Y->desc.cudnnDesc,
                Y->data->data,
                gamma->desc.cudnnDesc,
                gamma->data->data,
                beta->data->data,
                runningMean->data->data,
                runningVar->data->data,
                eps
                ))
                
        return Y;
    }
    
    cuTensorBase* batchnormOpGrads(cuTensorBase* X, cuTensorBase* Y, cuTensorBase* runningMean, cuTensorBase* runningVar,
                                   cuTensorBase* gamma, cuTensorBase* beta, float eps, float expAvgFactor){
        hipSetDevice(X->data->deviceID);
        
        float a = 1.0f, b = 1.0f;
        
        checkCUDNN(hipdnnBatchNormalizationBackward(
                cudnnHdlG,
                HIPDNN_BATCHNORM_SPATIAL,
                &a,
                &b,
                &a,
                &b,
                X->desc.cudnnDesc,
                X->data->data,
                Y->desc.cudnnDesc,
                Y->grad->data,
                X->desc.cudnnDesc,
                X->grad->data,
                gamma->desc.cudnnDesc,
                gamma->data->data,
                gamma->grad->data,
                beta->grad->data,
                eps,
                runningMean->grad->data,
                runningVar->grad->data
                ))
                
        return X;
    }
} // dylann