//
// Created by Dylan on 9/2/2022.
//

#include "cuActivation.cuh"

namespace dylann{
    hipdnnActivationDescriptor_t reluDescG;
    hipdnnActivationDescriptor_t sigmoidDescG;
    hipdnnActivationDescriptor_t tanhDescG;
    
    cuTensorBase* reluOp(cuTensorBase* X){
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&reluDescG);
            hipdnnSetActivationDescriptor(reluDescG, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                reluDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
                
        return X;
    }
    
    cuTensorBase* reluOp(cuTensorBase* X, cuTensorBase* Y){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&reluDescG);
            hipdnnSetActivationDescriptor(reluDescG, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                reluDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        return Y;
    }
    
    cuTensorBase* reluOpGrads(cuTensorBase* X, cuTensorBase* Y){
        float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           reluDescG,
                                           &alpha,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &beta,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
        return X;
    }
    
    
    cuTensorBase* sigmoidOp(cuTensorBase* X){
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&sigmoidDescG);
            hipdnnSetActivationDescriptor(sigmoidDescG, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                sigmoidDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
                
        return X;
    }
    
    cuTensorBase* sigmoidOp(cuTensorBase* X, cuTensorBase* Y){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&sigmoidDescG);
            hipdnnSetActivationDescriptor(sigmoidDescG, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                sigmoidDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        return Y;
    }
    
    cuTensorBase* sigmoidOpGrads(cuTensorBase* X, cuTensorBase* Y){
        float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           sigmoidDescG,
                                           &alpha,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &beta,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
        return X;
    }
    
    
    cuTensorBase* tanhOp(cuTensorBase* X){
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&tanhDescG);
            hipdnnSetActivationDescriptor(tanhDescG, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                tanhDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
                
        return X;
    }
    
    cuTensorBase* tanhOp(cuTensorBase* X, cuTensorBase* Y){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&reluDescG);
            hipdnnSetActivationDescriptor(tanhDescG, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                tanhDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        return Y;
    }
    
    cuTensorBase* tanhOpGrads(cuTensorBase* X, cuTensorBase* Y){
        float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           tanhDescG,
                                           &alpha,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &beta,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
        return X;
    }
    
    
    cuTensorBase* eluOp(cuTensorBase* X, float alpha){

        hipdnnActivationDescriptor_t eluDesc;
        hipdnnCreateActivationDescriptor(&eluDesc);
        hipdnnSetActivationDescriptor(eluDesc, HIPDNN_ACTIVATION_ELU, HIPDNN_NOT_PROPAGATE_NAN, alpha);
        
        
        float alpha2 = 1.0f, alpha3 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                eluDesc,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha3,
                X->desc.cudnnDesc,
                X->data->data
                ))
    
        hipdnnDestroyActivationDescriptor(eluDesc);
        return X;
    }
    
    cuTensorBase* eluOp(cuTensorBase* X, cuTensorBase* Y, float alpha){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        hipdnnActivationDescriptor_t eluDesc;
        hipdnnCreateActivationDescriptor(&eluDesc);
        hipdnnSetActivationDescriptor(eluDesc, HIPDNN_ACTIVATION_ELU, HIPDNN_NOT_PROPAGATE_NAN, alpha);
        
        
        float alpha2 = 1.0f, alpha3 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                eluDesc,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha3,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        hipdnnDestroyActivationDescriptor(eluDesc);
        return Y;
    }
    
    cuTensorBase* eluOpGrads(cuTensorBase* X, cuTensorBase* Y, float alpha){
        float a = 1.0f, beta = 0.0f;
    
        hipdnnActivationDescriptor_t eluDesc;
        hipdnnCreateActivationDescriptor(&eluDesc);
        hipdnnSetActivationDescriptor(eluDesc, HIPDNN_ACTIVATION_ELU, HIPDNN_NOT_PROPAGATE_NAN, alpha);
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           eluDesc,
                                           &a,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &beta,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
    
        hipdnnDestroyActivationDescriptor(eluDesc);
        return X;
    }
    
    cuTensorBase* swishOp(cuTensorBase* X, float beta){
        assertAllocated({X});
        assertOnSameDev({X});
        
        hipdnnActivationDescriptor_t swishDesc;
        hipdnnCreateActivationDescriptor(&swishDesc);
        hipdnnSetActivationDescriptor(swishDesc, HIPDNN_ACTIVATION_SWISH, HIPDNN_NOT_PROPAGATE_NAN, beta);
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                swishDesc,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
        hipdnnDestroyActivationDescriptor(swishDesc);
                
        return X;
    }
    
    cuTensorBase* swishOp(cuTensorBase* X, cuTensorBase* Y, float beta){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        hipdnnActivationDescriptor_t swishDesc;
        hipdnnCreateActivationDescriptor(&swishDesc);
        hipdnnSetActivationDescriptor(swishDesc, HIPDNN_ACTIVATION_SWISH, HIPDNN_NOT_PROPAGATE_NAN, beta);
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                swishDesc,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        hipdnnDestroyActivationDescriptor(swishDesc);
        return Y;
    }
    
    cuTensorBase* swishOpGrads(cuTensorBase* X, cuTensorBase* Y, float beta){
        float a = 1.0f, b = 0.0f;
    
        hipdnnActivationDescriptor_t swishDesc;
        hipdnnCreateActivationDescriptor(&swishDesc);
        hipdnnSetActivationDescriptor(swishDesc, HIPDNN_ACTIVATION_SWISH, HIPDNN_NOT_PROPAGATE_NAN, beta);
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           swishDesc,
                                           &a,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &b,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
    
        hipdnnDestroyActivationDescriptor(swishDesc);
        return X;
    }
    
    cuTensorBase* clippedReluOp(cuTensorBase* X, float ceiling){
        assertAllocated({X});
        assertOnSameDev({X});
        
        hipdnnActivationDescriptor_t clippedReluDesc;
        hipdnnCreateActivationDescriptor(&clippedReluDesc);
        hipdnnSetActivationDescriptor(clippedReluDesc, HIPDNN_ACTIVATION_CLIPPED_RELU, HIPDNN_NOT_PROPAGATE_NAN, ceiling);
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                clippedReluDesc,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
    
        hipdnnDestroyActivationDescriptor(clippedReluDesc);
        
        return X;
    }
    
    cuTensorBase* clippedReluOp(cuTensorBase* X, cuTensorBase* Y, float ceiling){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        hipdnnActivationDescriptor_t clippedReluDesc;
        hipdnnCreateActivationDescriptor(&clippedReluDesc);
        hipdnnSetActivationDescriptor(clippedReluDesc, HIPDNN_ACTIVATION_CLIPPED_RELU, HIPDNN_NOT_PROPAGATE_NAN, ceiling);
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                clippedReluDesc,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        hipdnnDestroyActivationDescriptor(clippedReluDesc);
        return Y;
    }
    
    cuTensorBase* clippedReluOpGrads(cuTensorBase* X, cuTensorBase* Y, float threshold){
        float a = 1.0f, b = 0.0f;
    
        hipdnnActivationDescriptor_t clippedReluDesc;
        hipdnnCreateActivationDescriptor(&clippedReluDesc);
        hipdnnSetActivationDescriptor(clippedReluDesc, HIPDNN_ACTIVATION_CLIPPED_RELU, HIPDNN_NOT_PROPAGATE_NAN, threshold);
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           clippedReluDesc,
                                           &a,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &b,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
    
        hipdnnDestroyActivationDescriptor(clippedReluDesc);
        return X;
    }
}