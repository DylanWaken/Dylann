//
// Created by Dylan on 9/2/2022.
//

#include "cuActivation.cuh"

namespace dylann{
    hipdnnActivationDescriptor_t reluDescG;
    hipdnnActivationDescriptor_t sigmoidDescG;
    hipdnnActivationDescriptor_t tanhDescG;
    
    cuTensorBase* reluOp(cuTensorBase* X, float alpha1, float alpha2){
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&reluDescG);
            hipdnnSetActivationDescriptor(reluDescG, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                reluDescG,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
                
        return X;
    }
    
    cuTensorBase* reluOp(cuTensorBase* X, cuTensorBase* Y, float alpha1, float alpha2){
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&reluDescG);
            hipdnnSetActivationDescriptor(reluDescG, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                reluDescG,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        return Y;
    }
    
    cuTensorBase* reluOpGrads(cuTensorBase* X, cuTensorBase* Y, float alpha1, float alpha2){
    
        //alpha1 = 1. alpha2 = 0.
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           reluDescG,
                                           &alpha1,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &alpha2,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
        return X;
    }
    
    
    cuTensorBase* sigmoidOp(cuTensorBase* X, float alpha1, float alpha2){
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&sigmoidDescG);
            hipdnnSetActivationDescriptor(sigmoidDescG, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
      //  float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                sigmoidDescG,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
                
        return X;
    }
    
    cuTensorBase* sigmoidOp(cuTensorBase* X, cuTensorBase* Y, float alpha1, float alpha2){
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&sigmoidDescG);
            hipdnnSetActivationDescriptor(sigmoidDescG, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        //float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                sigmoidDescG,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        return Y;
    }
    
    cuTensorBase* sigmoidOpGrads(cuTensorBase* X, cuTensorBase* Y, float alpha1, float alpha2){
        //float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           sigmoidDescG,
                                           &alpha1,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &alpha2,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
        return X;
    }
    
    
    cuTensorBase* tanhOp(cuTensorBase* X, float alpha1, float alpha2){
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&tanhDescG);
            hipdnnSetActivationDescriptor(tanhDescG, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
       // float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                tanhDescG,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
                
        return X;
    }
    
    cuTensorBase* tanhOp(cuTensorBase* X, cuTensorBase* Y, float alpha1, float alpha2){
        
        
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&reluDescG);
            hipdnnSetActivationDescriptor(tanhDescG, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        //float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                tanhDescG,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        return Y;
    }
    
    cuTensorBase* tanhOpGrads(cuTensorBase* X, cuTensorBase* Y, float alpha1, float alpha2){
        //float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           tanhDescG,
                                           &alpha1,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &alpha2,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
        return X;
    }
    
    
    cuTensorBase* eluOp(cuTensorBase* X, float alpha, float alpha1, float alpha2){

        hipdnnActivationDescriptor_t eluDesc;
        hipdnnCreateActivationDescriptor(&eluDesc);
        hipdnnSetActivationDescriptor(eluDesc, HIPDNN_ACTIVATION_ELU, HIPDNN_NOT_PROPAGATE_NAN, alpha);
        
        
        //float alpha2 = 1.0f, alpha3 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                eluDesc,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
    
        hipdnnDestroyActivationDescriptor(eluDesc);
        return X;
    }
    
    cuTensorBase* eluOp(cuTensorBase* X, cuTensorBase* Y, float alpha, float alpha1, float alpha2){
        
        
        
        hipdnnActivationDescriptor_t eluDesc;
        hipdnnCreateActivationDescriptor(&eluDesc);
        hipdnnSetActivationDescriptor(eluDesc, HIPDNN_ACTIVATION_ELU, HIPDNN_NOT_PROPAGATE_NAN, alpha);
        
        
        //float alpha2 = 1.0f, alpha3 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                eluDesc,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        hipdnnDestroyActivationDescriptor(eluDesc);
        return Y;
    }
    
    cuTensorBase* eluOpGrads(cuTensorBase* X, cuTensorBase* Y, float alpha, float alpha1, float alpha2){
        //float a = 1.0f, beta = 0.0f;
    
        hipdnnActivationDescriptor_t eluDesc;
        hipdnnCreateActivationDescriptor(&eluDesc);
        hipdnnSetActivationDescriptor(eluDesc, HIPDNN_ACTIVATION_ELU, HIPDNN_NOT_PROPAGATE_NAN, alpha);
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           eluDesc,
                                           &alpha1,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &alpha2,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
    
        hipdnnDestroyActivationDescriptor(eluDesc);
        return X;
    }
    
    cuTensorBase* swishOp(cuTensorBase* X, float beta, float alpha1, float alpha2){
        
        hipdnnActivationDescriptor_t swishDesc;
        hipdnnCreateActivationDescriptor(&swishDesc);
        hipdnnSetActivationDescriptor(swishDesc, HIPDNN_ACTIVATION_SWISH, HIPDNN_NOT_PROPAGATE_NAN, beta);
        
        //float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                swishDesc,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
        hipdnnDestroyActivationDescriptor(swishDesc);
                
        return X;
    }
    
    cuTensorBase* swishOp(cuTensorBase* X, cuTensorBase* Y, float beta, float alpha1, float alpha2){
        
        hipdnnActivationDescriptor_t swishDesc;
        hipdnnCreateActivationDescriptor(&swishDesc);
        hipdnnSetActivationDescriptor(swishDesc, HIPDNN_ACTIVATION_SWISH, HIPDNN_NOT_PROPAGATE_NAN, beta);
        
        //float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                swishDesc,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        hipdnnDestroyActivationDescriptor(swishDesc);
        return Y;
    }
    
    cuTensorBase* swishOpGrads(cuTensorBase* X, cuTensorBase* Y, float beta, float alpha1, float alpha2){
        //float a = 1.0f, b = 0.0f;
    
        hipdnnActivationDescriptor_t swishDesc;
        hipdnnCreateActivationDescriptor(&swishDesc);
        hipdnnSetActivationDescriptor(swishDesc, HIPDNN_ACTIVATION_SWISH, HIPDNN_NOT_PROPAGATE_NAN, beta);
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           swishDesc,
                                           &alpha1,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &alpha2,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
    
        hipdnnDestroyActivationDescriptor(swishDesc);
        return X;
    }
    
    cuTensorBase* clippedReluOp(cuTensorBase* X, float ceiling, float alpha1, float alpha2){
        
        hipdnnActivationDescriptor_t clippedReluDesc;
        hipdnnCreateActivationDescriptor(&clippedReluDesc);
        hipdnnSetActivationDescriptor(clippedReluDesc, HIPDNN_ACTIVATION_CLIPPED_RELU, HIPDNN_NOT_PROPAGATE_NAN, ceiling);
        
        //float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                clippedReluDesc,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
    
        hipdnnDestroyActivationDescriptor(clippedReluDesc);
        
        return X;
    }
    
    cuTensorBase* clippedReluOp(cuTensorBase* X, cuTensorBase* Y, float ceiling, float alpha1, float alpha2){
        
        
        
        hipdnnActivationDescriptor_t clippedReluDesc;
        hipdnnCreateActivationDescriptor(&clippedReluDesc);
        hipdnnSetActivationDescriptor(clippedReluDesc, HIPDNN_ACTIVATION_CLIPPED_RELU, HIPDNN_NOT_PROPAGATE_NAN, ceiling);
        
        //float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                clippedReluDesc,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        hipdnnDestroyActivationDescriptor(clippedReluDesc);
        return Y;
    }
    
    cuTensorBase* clippedReluOpGrads(cuTensorBase* X, cuTensorBase* Y, float threshold, float alpha1, float alpha2){
        //float a = 1.0f, b = 0.0f;
    
        hipdnnActivationDescriptor_t clippedReluDesc;
        hipdnnCreateActivationDescriptor(&clippedReluDesc);
        hipdnnSetActivationDescriptor(clippedReluDesc, HIPDNN_ACTIVATION_CLIPPED_RELU, HIPDNN_NOT_PROPAGATE_NAN, threshold);
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                           clippedReluDesc,
                                           &alpha1,
                                           Y->desc.cudnnDesc,
                                           Y->data->data,
                                           Y->desc.cudnnDesc,
                                           Y->grad->data,
                                           X->desc.cudnnDesc,
                                           X->data->data,
                                           &alpha2,
                                           X->desc.cudnnDesc,
                                           X->grad->data))
    
        hipdnnDestroyActivationDescriptor(clippedReluDesc);
        return X;
    }
}