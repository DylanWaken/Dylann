//
// Created by Dylan on 9/2/2022.
//

#include "cuActivation.cuh"

namespace dylann{
    hipdnnActivationDescriptor_t reluDescG;
    hipdnnActivationDescriptor_t sigmoidDescG;
    hipdnnActivationDescriptor_t tanhDescG;
    
    cuTensorBase* reluOp(cuTensorBase* X){
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&reluDescG);
            hipdnnSetActivationDescriptor(reluDescG, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                reluDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
                
        return X;
    }
    
    cuTensorBase* reluOp(cuTensorBase* X, cuTensorBase* Y){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&reluDescG);
            hipdnnSetActivationDescriptor(reluDescG, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                reluDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        return Y;
    }
    
    cuTensorBase* sigmoidOp(cuTensorBase* X){
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&sigmoidDescG);
            hipdnnSetActivationDescriptor(sigmoidDescG, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                sigmoidDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
                
        return X;
    }
    
    cuTensorBase* sigmoidOp(cuTensorBase* X, cuTensorBase* Y){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&sigmoidDescG);
            hipdnnSetActivationDescriptor(sigmoidDescG, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                sigmoidDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        return Y;
    }
    
    cuTensorBase* tanhOp(cuTensorBase* X){
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&tanhDescG);
            hipdnnSetActivationDescriptor(tanhDescG, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                tanhDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
                
        return X;
    }
    
    cuTensorBase* tanhOp(cuTensorBase* X, cuTensorBase* Y){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        if(reluDescG == nullptr){
            hipdnnCreateActivationDescriptor(&reluDescG);
            hipdnnSetActivationDescriptor(tanhDescG, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 0);
        }
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                tanhDescG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        return Y;
    }
    
    cuTensorBase* eluOp(cuTensorBase* X, float alpha){

        hipdnnActivationDescriptor_t eluDesc;
        hipdnnCreateActivationDescriptor(&eluDesc);
        hipdnnSetActivationDescriptor(eluDesc, HIPDNN_ACTIVATION_ELU, HIPDNN_NOT_PROPAGATE_NAN, alpha);
        
        
        float alpha2 = 1.0f, alpha3 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                eluDesc,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha3,
                X->desc.cudnnDesc,
                X->data->data
                ))
    
        hipdnnDestroyActivationDescriptor(eluDesc);
        return X;
    }
    
    cuTensorBase* eluOp(cuTensorBase* X, cuTensorBase* Y, float alpha){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        hipdnnActivationDescriptor_t eluDesc;
        hipdnnCreateActivationDescriptor(&eluDesc);
        hipdnnSetActivationDescriptor(eluDesc, HIPDNN_ACTIVATION_ELU, HIPDNN_NOT_PROPAGATE_NAN, alpha);
        
        
        float alpha2 = 1.0f, alpha3 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                eluDesc,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha3,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        hipdnnDestroyActivationDescriptor(eluDesc);
        return Y;
    }
    
    cuTensorBase* swishOp(cuTensorBase* X, float beta){
        assertAllocated({X});
        assertOnSameDev({X});
        
        hipdnnActivationDescriptor_t swishDesc;
        hipdnnCreateActivationDescriptor(&swishDesc);
        hipdnnSetActivationDescriptor(swishDesc, HIPDNN_ACTIVATION_SWISH, HIPDNN_NOT_PROPAGATE_NAN, beta);
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                swishDesc,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
        hipdnnDestroyActivationDescriptor(swishDesc);
                
        return X;
    }
    
    cuTensorBase* swishOp(cuTensorBase* X, cuTensorBase* Y, float beta){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        hipdnnActivationDescriptor_t swishDesc;
        hipdnnCreateActivationDescriptor(&swishDesc);
        hipdnnSetActivationDescriptor(swishDesc, HIPDNN_ACTIVATION_SWISH, HIPDNN_NOT_PROPAGATE_NAN, beta);
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                swishDesc,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        hipdnnDestroyActivationDescriptor(swishDesc);
        return Y;
    }
    
    cuTensorBase* clippedReluOp(cuTensorBase* X, float ceiling){
        assertAllocated({X});
        assertOnSameDev({X});
        
        hipdnnActivationDescriptor_t clippedReluDesc;
        hipdnnCreateActivationDescriptor(&clippedReluDesc);
        hipdnnSetActivationDescriptor(clippedReluDesc, HIPDNN_ACTIVATION_CLIPPED_RELU, HIPDNN_NOT_PROPAGATE_NAN, ceiling);
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                clippedReluDesc,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->data->data
                ))
    
        hipdnnDestroyActivationDescriptor(clippedReluDesc);
        
        return X;
    }
    
    cuTensorBase* clippedReluOp(cuTensorBase* X, cuTensorBase* Y, float ceiling){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        hipdnnActivationDescriptor_t clippedReluDesc;
        hipdnnCreateActivationDescriptor(&clippedReluDesc);
        hipdnnSetActivationDescriptor(clippedReluDesc, HIPDNN_ACTIVATION_CLIPPED_RELU, HIPDNN_NOT_PROPAGATE_NAN, ceiling);
        
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(hipdnnActivationForward(
                cudnnHdlG,
                clippedReluDesc,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
                
        hipdnnDestroyActivationDescriptor(clippedReluDesc);
        return Y;
    }
    
    void GRAD_RELU::backwardCalc(dylann::cuTensorBase *current) {
        
        cout<<"GRAD_RELU::backwardCalc"<<endl;
        float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                reluDescG,
                                &alpha,
                                current->desc.cudnnDesc,
                                current->data->data,
                                current->desc.cudnnDesc,
                                current->grad->data,
                                X->desc.cudnnDesc,
                                X->data->data,
                                &beta,
                                X->desc.cudnnDesc,
                                X->grad->data))
    }
    
    void GRAD_SIGMOID::backwardCalc(dylann::cuTensorBase *current) {
        cout<<"GRAD_SIGMOID::backwardCalc"<<endl;
        float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                sigmoidDescG,
                                &alpha,
                                current->desc.cudnnDesc,
                                current->data->data,
                                current->desc.cudnnDesc,
                                current->grad->data,
                                X->desc.cudnnDesc,
                                X->data->data,
                                &beta,
                                X->desc.cudnnDesc,
                                X->grad->data))
    }
    
    void GRAD_TANH::backwardCalc(dylann::cuTensorBase *current) {
        cout<<"GRAD_TANH::backwardCalc"<<endl;
        float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                tanhDescG,
                                &alpha,
                                current->desc.cudnnDesc,
                                current->data->data,
                                current->desc.cudnnDesc,
                                current->grad->data,
                                X->desc.cudnnDesc,
                                X->data->data,
                                &beta,
                                X->desc.cudnnDesc,
                                X->grad->data))
    }
    
    void GRAD_ELU::backwardCalc(dylann::cuTensorBase *current) {
        cout<<"GRAD_ELU::backwardCalc"<<endl;
        float a = 1.0f, beta = 0.0f;
        
        hipdnnActivationDescriptor_t eluDesc;
        hipdnnCreateActivationDescriptor(&eluDesc);
        hipdnnSetActivationDescriptor(eluDesc, HIPDNN_ACTIVATION_ELU, HIPDNN_NOT_PROPAGATE_NAN, alpha);
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                eluDesc,
                                &a,
                                current->desc.cudnnDesc,
                                current->data->data,
                                current->desc.cudnnDesc,
                                current->grad->data,
                                X->desc.cudnnDesc,
                                X->data->data,
                                &beta,
                                X->desc.cudnnDesc,
                                X->grad->data))
                                
        hipdnnDestroyActivationDescriptor(eluDesc);
    }
    
    void GRAD_SWISH::backwardCalc(dylann::cuTensorBase *current) {
        cout<<"GRAD_SWISH::backwardCalc"<<endl;
        float a = 1.0f, b = 0.0f;
        
        hipdnnActivationDescriptor_t swishDesc;
        hipdnnCreateActivationDescriptor(&swishDesc);
        hipdnnSetActivationDescriptor(swishDesc, HIPDNN_ACTIVATION_SWISH, HIPDNN_NOT_PROPAGATE_NAN, beta);
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                swishDesc,
                                &a,
                                current->desc.cudnnDesc,
                                current->data->data,
                                current->desc.cudnnDesc,
                                current->grad->data,
                                X->desc.cudnnDesc,
                                X->data->data,
                                &b,
                                X->desc.cudnnDesc,
                                X->grad->data))
                                
        hipdnnDestroyActivationDescriptor(swishDesc);
    }
    
    void GRAD_CLIPPED_RELU::backwardCalc(dylann::cuTensorBase *current) {
        cout<<"GRAD_CLIPPED_RELU::backwardCalc"<<endl;
        float a = 1.0f, b = 0.0f;
        
        hipdnnActivationDescriptor_t clippedReluDesc;
        hipdnnCreateActivationDescriptor(&clippedReluDesc);
        hipdnnSetActivationDescriptor(clippedReluDesc, HIPDNN_ACTIVATION_CLIPPED_RELU, HIPDNN_NOT_PROPAGATE_NAN, threshold);
    
        checkCUDNN(hipdnnActivationBackward(cudnnHdlG,
                                clippedReluDesc,
                                &a,
                                current->desc.cudnnDesc,
                                current->data->data,
                                current->desc.cudnnDesc,
                                current->grad->data,
                                X->desc.cudnnDesc,
                                X->data->data,
                                &b,
                                X->desc.cudnnDesc,
                                X->grad->data))
                                
        hipdnnDestroyActivationDescriptor(clippedReluDesc);
    }
}