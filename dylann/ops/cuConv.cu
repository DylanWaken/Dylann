//
// Created by Dylan on 8/31/2022.
//

#include "cuConv.cuh"
namespace dylann{
    cuTensorBase* conv2dOp(cuTensorBase* X, cuTensorBase* W, cuTensorBase* B, cuTensorBase* Y,
                           int padH, int padW, int strideH, int strideW, int dilationH, int dilationW){
        assertAllocated({W, B, X, Y});
        assertOnSameDev({W, B, X, Y});
    
        hipSetDevice(W->data->deviceID);
    
        hipdnnConvolutionDescriptor_t convDesc;
        hipdnnCreateConvolutionDescriptor(&convDesc);
        hipdnnFilterDescriptor_t filterDesc;
        hipdnnCreateFilterDescriptor(&filterDesc);
        hipdnnActivationDescriptor_t activationDesc;
        hipdnnCreateActivationDescriptor(&activationDesc);
    
        checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padH, padW, strideH, strideW, dilationH, dilationW,
                                        HIPDNN_CONVOLUTION, X->desc.dType))
        checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc, X->desc.dType,
                                   HIPDNN_TENSOR_NCHW,
                                   (int)W->desc.sizes.n,
                                   (int)W->desc.sizes.c,
                                   (int)W->desc.sizes.h,
                                   (int)W->desc.sizes.w
                                   ))
                                   
        checkCUDNN(hipdnnSetActivationDescriptor(activationDesc, HIPDNN_ACTIVATION_PATHTRU, HIPDNN_NOT_PROPAGATE_NAN, 0))
    
        checkCUDNN(hipdnnSetConvolutionMathType(convDesc,  HIPDNN_TENSOR_OP_MATH))
    
        float alpha = 1.0f, alpha2 = 0.0f;

        checkCUDNN(cudnnConvolutionBiasActivationForward(
                cudnnHdlG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                filterDesc,
                W->data->data,
                convDesc,
                HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM,
                cudnnWorkspaceG,
                CUDNN_WORKSPACE_SIZE_G,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data,
                B->desc.cudnnDesc,
                B->data->data,
                activationDesc,
                Y->desc.cudnnDesc,
                Y->data->data
                ))

        hipdnnDestroyActivationDescriptor(activationDesc);
        hipdnnDestroyFilterDescriptor(filterDesc);
        hipdnnDestroyConvolutionDescriptor(convDesc);
        hipMemset(cudnnWorkspaceG, 0, CUDNN_WORKSPACE_SIZE_G);
    
        assertCuda(__FILE__, __LINE__);
        return Y;
    }
    
    cuTensorBase* conv2dActiveOp(cuTensorBase* X, cuTensorBase* W, cuTensorBase* B, cuTensorBase* Y,
                                 int padH, int padW, int strideH, int strideW, int dilationH, int dilationW,
                                 hipdnnActivationMode_t mode, float coef){
        assertAllocated({W, B, X, Y});
        assertOnSameDev({W, B, X, Y});
    
        hipSetDevice(W->data->deviceID);
    
        hipdnnConvolutionDescriptor_t convDesc;
        hipdnnCreateConvolutionDescriptor(&convDesc);
        hipdnnFilterDescriptor_t filterDesc;
        hipdnnCreateFilterDescriptor(&filterDesc);
        hipdnnActivationDescriptor_t activationDesc;
        hipdnnCreateActivationDescriptor(&activationDesc);
    
        checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padH, padW, strideH, strideW, dilationH, dilationW,
                                                   HIPDNN_CONVOLUTION, X->desc.dType))
        checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc, X->desc.dType,
                                              HIPDNN_TENSOR_NCHW,
                                              (int)W->desc.sizes.n,
                                              (int)W->desc.sizes.c,
                                              (int)W->desc.sizes.h,
                                              (int)W->desc.sizes.w))
        checkCUDNN(hipdnnSetActivationDescriptor(activationDesc, mode, HIPDNN_NOT_PROPAGATE_NAN, coef))
    
        checkCUDNN(hipdnnSetConvolutionMathType(convDesc,  HIPDNN_TENSOR_OP_MATH))
    
        float alpha = 1.0f, alpha2 = 0.0f;
    
        checkCUDNN(cudnnConvolutionBiasActivationForward(
                cudnnHdlG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                filterDesc,
                W->data->data,
                convDesc,
                HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM,
                cudnnWorkspaceG,
                CUDNN_WORKSPACE_SIZE_G,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data,
                B->desc.cudnnDesc,
                B->data->data,
                activationDesc,
                Y->desc.cudnnDesc,
                Y->data->data
        ))
    
        hipdnnDestroyActivationDescriptor(activationDesc);
        hipdnnDestroyFilterDescriptor(filterDesc);
        hipdnnDestroyConvolutionDescriptor(convDesc);
        hipMemset(cudnnWorkspaceG, 0, CUDNN_WORKSPACE_SIZE_G);
        
        return Y;
    }
    
    cuTensorBase* conv2dOpGrads(cuTensorBase* X, cuTensorBase* W, cuTensorBase* B, cuTensorBase* Y,
                                int padH, int padW, int strideH, int strideW, int dilationH, int dilationW){
        hipSetDevice(W->data->deviceID);
    
        hipdnnConvolutionDescriptor_t convDesc;
        hipdnnCreateConvolutionDescriptor(&convDesc);
    
        hipdnnFilterDescriptor_t filterDesc;
        hipdnnCreateFilterDescriptor(&filterDesc);
    
        float alpha = 1.0f, beta = 1.0f;
    
        checkCUDNN(hipdnnSetConvolution2dDescriptor(convDesc, padH, padW, strideH, strideW, dilationH, dilationW,
                                                   HIPDNN_CONVOLUTION, Y->desc.dType))
        checkCUDNN(hipdnnSetFilter4dDescriptor(filterDesc, W->desc.dType,
                                              HIPDNN_TENSOR_NCHW,
                                              (int)W->desc.sizes.n,
                                              (int)W->desc.sizes.c,
                                              (int)W->desc.sizes.h,
                                              (int)W->desc.sizes.w));
    
        checkCUDNN(hipdnnSetConvolutionMathType(convDesc,  HIPDNN_TENSOR_OP_MATH))
    
        checkCUDNN(hipdnnConvolutionBackwardData(cudnnHdlG,
                                                &alpha,
                                                filterDesc,
                                                W->data->data,
                                                Y->desc.cudnnDesc,
                                                Y->grad->data,
                                                convDesc,
                                                HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
                                                cudnnWorkspaceG,
                                                CUDNN_WORKSPACE_SIZE_G,
                                                &beta,
                                                X->desc.cudnnDesc,
                                                X->grad->data
        ))
    
        checkCUDNN(hipdnnConvolutionBackwardFilter(
                cudnnHdlG,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                Y->desc.cudnnDesc,
                Y->grad->data,
                convDesc,
                HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                cudnnWorkspaceG,
                CUDNN_WORKSPACE_SIZE_G,
                &beta,
                filterDesc,
                W->grad->data
        ))
    
        checkCUDNN(hipdnnConvolutionBackwardBias(
                cudnnHdlG,
                &alpha,
                Y->desc.cudnnDesc,
                Y->grad->data,
                &beta,
                B->desc.cudnnDesc,
                B->grad->data
        ))
    
        hipdnnDestroyFilterDescriptor(filterDesc);
        hipdnnDestroyConvolutionDescriptor(convDesc);
        return X;
    }
}
