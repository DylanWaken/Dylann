#include "opRegistry.cuh"

namespace dylann{
    cuTensorBase* copy(cuTensorBase* A, cuTensorBase*B){
        assertAllocated({A, B});
        hipSetDevice(A->data->deviceID);
        
        hipMemcpy(B->data->data, A->data->data, A->data->memSize, hipMemcpyDeviceToDevice);
        assertCuda(__FILE__, __LINE__);
        return B;
    }
    
    cuTensorBase* mergeGradBuf(cuTensorBase* A){
        assertAllocated({A});
        assert(A->desc.withGradBuf);
        hipSetDevice(A->data->deviceID);
        
        int a = 1, b = 1;
        checkCUDNN(hipdnnAddTensor(cudnnHdlG,
                                  &a,
                                  A->desc.cudnnDesc,
                                  A->grad->data,
                                  &b,
                                  A->desc.cudnnDesc,
                                  A->gradBuf->data
        ));
        return A;
    }
    
    void GRAD_ADD_A::backward(cuTensorBase *current) {
        assert(current->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                    current->desc.cudnnDesc,
                                    current->grad->data,
                                    &alpha));
    }
    
    void GRAD_ADD_B::backward(cuTensorBase *current) {
        assert(prev->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        if (prev->desc.withGradBuf){
            hipMemcpy(prev->gradBuf->data, current->grad->data, prev->grad->memSize, hipMemcpyDeviceToDevice);
            assertCuda(__FILE__, __LINE__);
        
            checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                        prev->desc.cudnnDesc,
                                        prev->gradBuf->data,
                                        &beta))
            mergeGradBuf(prev);
            return;
        }
    
        hipMemcpy(prev->grad->data, current->grad->data, current->grad->memSize, hipMemcpyDeviceToDevice);
        assertCuda(__FILE__, __LINE__);
    
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                    prev->desc.cudnnDesc,
                                    prev->grad->data,
                                    &beta))
    }
    
    void GRAD_SCALE::backward(cuTensorBase *current) {
        assert(current->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                    current->desc.cudnnDesc,
                                    current->grad->data,
                                    &alpha));
    }
}