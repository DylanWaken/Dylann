//
// Created by Dylan on 8/25/2022.
//

#include "cuLinear.cuh"

namespace dylann{
    
    void fillBias(cuTensorBase* B, cuTensorBase* Y){
        for(auto i = 0; i < Y->desc.sizes.h; i++){
            auto offset = i * Y->desc.sizes.w;
            char* destPtr = (char*)Y->data->data + offset * Y->desc.elementSize;
            hipMemcpy(destPtr, B->data->data, B->data->memSize, hipMemcpyDeviceToDevice);
        }
        assertCuda(__FILE__, __LINE__);
    };
    
    void FLOAT_LINEAR(cuTensorBase* W, cuTensorBase* X, cuTensorBase* Y){
        //run gemm (linear operation)
        float a = 1.0f, b = 1.0f;
    
        checkCUBLAS(hipblasSgemm(cublasHdlG,
                                   HIPBLAS_OP_T,   //row major to column major for weights
                                   HIPBLAS_OP_N,   //read the original row major as column major, auto trans
                                   Y->desc.sizes.w,
                                   X->desc.sizes.h,
                                   X->desc.sizes.w,
                                   &a,
                                   (float*)W->data->data,
                                   W->desc.sizes.h,
                                   (float*)X->data->data,
                                   X->desc.sizes.w,
                                   &b,
                                   (float*)Y->data->data,
                                   Y->desc.sizes.w
        ))
    }
    
    void FLOAT_LINEAR_GRAD_X(cuTensorBase* W, cuTensorBase* X, cuTensorBase* Y){
        float a = 1.0f, b = 1.0f;
    
        checkCUBLAS(hipblasSgemm(cublasHdlG,
                                   HIPBLAS_OP_N,  //auto transpose for weights
                                   HIPBLAS_OP_N,
                                   W->desc.sizes.h,
                                   Y->desc.sizes.h,
                                   W->desc.sizes.w,
                                   &a,
                                   (float*)W->data->data,
                                   W->desc.sizes.h,
                                   (float*)Y->grad->data,
                                   Y->desc.sizes.w,
                                   &b,
                                   (float*)X->grad->data,
                                   X->desc.sizes.w
        ))
    }
    
    void FLOAT_LINEAR_GRAD_W(cuTensorBase* W, cuTensorBase* X, cuTensorBase* Y){
        float a = 1.0f, b = 0.0f;
    
        checkCUBLAS(hipblasSgemm(cublasHdlG,
                                   HIPBLAS_OP_T,
                                   HIPBLAS_OP_T,  //we need to recover the "row major"
                                   Y->desc.sizes.w,
                                   X->desc.sizes.w,
                                   X->desc.sizes.h,
                                   &a,
                                   (float*)Y->grad->data,
                                   Y->desc.sizes.h,
                                   (float*)X->data->data,
                                   X->desc.sizes.w,
                                  &b,
                                  (float*)W->grad->data,
                                  W->desc.sizes.w
                                   ))
    }
    
    void HALF_LINEAR(cuTensorBase* W, cuTensorBase* X, cuTensorBase* Y){
        //run gemm (linear operation)
        half a = 1.0, b = 1.0;
    
        checkCUBLAS(hipblasHgemm(cublasHdlG,
                                   HIPBLAS_OP_T,   //row major to column major for weights
                                   HIPBLAS_OP_N,   //read the original row major as  column major, auto trans
                                      W->desc.sizes.w,
                                        X->desc.sizes.h,
                                        X->desc.sizes.w,
                                        &a,
                                        (half*)W->data->data,
                                        W->desc.sizes.h,
                                        (half*)X->data->data,
                                        X->desc.sizes.w,
                                        &b,
                                        (half*)Y->data->data,
                                        Y->desc.sizes.w
        ))
    }
    
    void HALF_LINEAR_GRAD_X(cuTensorBase* W, cuTensorBase* X, cuTensorBase* Y){
        half a = 1.0, b = 1.0;
    
        checkCUBLAS(hipblasHgemm(cublasHdlG,
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                W->desc.sizes.h,
                                Y->desc.sizes.h,
                                W->desc.sizes.w,
                                &a,
                                (half*)W->data->data,
                                W->desc.sizes.h,
                                (half*)Y->grad->data,
                                Y->desc.sizes.w,
                                &b,
                                (half*)X->grad->data,
                                X->desc.sizes.w
        ))
    }
    
    void HALF_LINEAR_GRAD_W(cuTensorBase* W, cuTensorBase* X, cuTensorBase* Y){
        half a = 1.0, b = 0.0;
    
        checkCUBLAS(hipblasHgemm(cublasHdlG,
                                HIPBLAS_OP_T,
                                HIPBLAS_OP_T,
                                Y->desc.sizes.w,
                                X->desc.sizes.w,
                                X->desc.sizes.h,
                                &a,
                                (half*)Y->grad->data,
                                Y->desc.sizes.h,
                                (half*)X->data->data,
                                X->desc.sizes.w,
                                &b,
                                (half*)W->grad->data,
                                W->desc.sizes.w
        ))
    }
    
    void DOUBLE_LINEAR(cuTensorBase* W, cuTensorBase* X, cuTensorBase* Y){
        //run gemm (linear operation)
        double a = 1.0, b = 1.0;
    
        checkCUBLAS(hipblasDgemm(cublasHdlG,
                                   HIPBLAS_OP_T,   //row major to column major for weights
                                   HIPBLAS_OP_N,   //read the original row major as column major, auto trans
                                   W->desc.sizes.w,
                                   X->desc.sizes.h,
                                   X->desc.sizes.w,
                                   &a,
                                   (double*)W->data->data,
                                   W->desc.sizes.h,
                                   (double*)X->data->data,
                                   X->desc.sizes.w,
                                   &b,
                                   (double*)Y->data->data,
                                   Y->desc.sizes.w))
    }
    
    void DOUBLE_LINEAR_GRAD_X(cuTensorBase* W, cuTensorBase* X, cuTensorBase* Y){
        double a = 1.0, b = 1.0;
    
        checkCUBLAS(hipblasDgemm(cublasHdlG,
                                   HIPBLAS_OP_N,
                                   HIPBLAS_OP_N,
                                   W->desc.sizes.h,
                                   Y->desc.sizes.h,
                                   W->desc.sizes.w,
                                   &a,
                                   (double*)W->data->data,
                                   W->desc.sizes.h,
                                   (double*)Y->grad->data,
                                   Y->desc.sizes.w,
                                   &b,
                                   (double*)X->grad->data,
                                   X->desc.sizes.w
        ))
    }
    
    void DOUBLE_LINEAR_GRAD_W(cuTensorBase* W, cuTensorBase* X, cuTensorBase* Y){
        double a = 1.0, b = 0.0;
    
        checkCUBLAS(hipblasDgemm(cublasHdlG,
                                   HIPBLAS_OP_T,
                                   HIPBLAS_OP_T,
                                   Y->desc.sizes.w,
                                   X->desc.sizes.w,
                                   X->desc.sizes.h,
                                   &a,
                                   (double*)Y->grad->data,
                                   Y->desc.sizes.h,
                                   (double*)X->data->data,
                                   X->desc.sizes.w,
                                   &b,
                                   (double*)W->grad->data,
                                   W->desc.sizes.w
        ))
    }
    
    cuTensorBase *linearOp(cuTensorBase* W, cuTensorBase* B, cuTensorBase* X, cuTensorBase* Y){
        //set cublas
        checkCUBLAS(hipblasSetMathMode(cublasHdlG, HIPBLAS_TENSOR_OP_MATH))
        fillBias(B, Y);
        
        //assert same dtye
        assert(W->desc.dType == X->desc.dType
           && W->desc.dType == Y->desc.dType);
        
        //run gemm (linear operation)
        switch (X->desc.dType) {
            case HIPDNN_DATA_FLOAT:
                FLOAT_LINEAR(W, X, Y);
                break;
            case HIPDNN_DATA_HALF:
                HALF_LINEAR(W, X, Y);
                break;
            case HIPDNN_DATA_DOUBLE:
                DOUBLE_LINEAR(W, X, Y);
                break;
            default:
                throw std::runtime_error("unsupported dtype");
        }
        
        return Y;
    }
    
    void GRAD_LINEAR::backward(cuTensorBase *current) {
        checkCUBLAS(hipblasSetMathMode(cublasHdlG, HIPBLAS_TENSOR_OP_MATH))
    
        //assert same dtype
        assert(current->desc.dType == X->desc.dType
               && current->desc.dType == current->desc.dType);
        
        cout<<"GRAD LINEAR CALLED"<<endl;
        
        //run gradient for features
        switch (X->desc.dType) {
            case HIPDNN_DATA_FLOAT:
                FLOAT_LINEAR_GRAD_X(W, X, current);
                break;
            case HIPDNN_DATA_HALF:
                HALF_LINEAR_GRAD_X(W, X, current);
                break;
            case HIPDNN_DATA_DOUBLE:
                DOUBLE_LINEAR_GRAD_X(W, X, current);
                break;
            default:
                throw std::runtime_error("unsupported dtype");
        }
        
        //run gradients for weights
        switch (X->desc.dType) {
            case HIPDNN_DATA_FLOAT:
                FLOAT_LINEAR_GRAD_W(W, X, current);
                break;
            case HIPDNN_DATA_HALF:
                HALF_LINEAR_GRAD_W(W, X, current);
                break;
            case HIPDNN_DATA_DOUBLE:
                DOUBLE_LINEAR_GRAD_W(W, X, current);
                break;
            default:
                throw std::runtime_error("unsupported dtype");
        }
        
        //run gradients for biases
        hipMemcpy(B->grad->data, current->grad->data, B->grad->memSize, hipMemcpyDeviceToHost);
    }
}