//
// Created by Dylan on 9/4/2022.
//

#include "cuPool.cuh"

namespace dylann {
    cuTensorBase *maxPoolOp(cuTensorBase* X, cuTensorBase* Y, int rangeH, int rangeW,
                            int padH, int padW, int strideH, int strideW){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        hipSetDevice(X->data->deviceID);
        
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_MAX,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    rangeH,
                                    rangeW,
                                    padH,
                                    padW,
                                    strideH,
                                    strideW);
    
        float alpha = 1.0f, beta = 0.0f;
        
        checkCUDNN(hipdnnPoolingForward(
                cudnnHdlG,
                poolDesc,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &beta,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
    
        hipdnnDestroyPoolingDescriptor(poolDesc);
        
        return Y;
    }
    
    cuTensorBase *maxPoolOpGrads(cuTensorBase* X, cuTensorBase* Y,  int rangeH, int rangeW,
                                 int padH, int padW, int strideH, int strideW){
        hipSetDevice(X->data->deviceID);
    
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_MAX,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    rangeH,
                                    rangeW,
                                    padH,
                                    padW,
                                    strideH,
                                    strideW);
        
        float alpha = 1.0f, beta = 1.0f;
        
        checkCUDNN(hipdnnPoolingBackward(
                cudnnHdlG,
                poolDesc,
                &alpha,
                Y->desc.cudnnDesc,
                Y->data->data,
                Y->desc.cudnnDesc,
                Y->grad->data,
                X->desc.cudnnDesc,
                X->data->data,
                &beta,
                X->desc.cudnnDesc,
                X->grad->data
                ))
        
        hipdnnDestroyPoolingDescriptor(poolDesc);
        
        return X;
    }
    
    cuTensorBase *avgPoolOp(cuTensorBase* X, cuTensorBase* Y, int rangeH, int rangeW,
                            int padH, int padW, int strideH, int strideW){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
    
        hipSetDevice(X->data->deviceID);
    
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    rangeH,
                                    rangeW,
                                    padH,
                                    padW,
                                    strideH,
                                    strideW);
    
        float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnPoolingForward(
                cudnnHdlG,
                poolDesc,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &beta,
                Y->desc.cudnnDesc,
                Y->data->data
        ))
    
        hipdnnDestroyPoolingDescriptor(poolDesc);
    
        return Y;
    }
    
    cuTensorBase *avgPoolOpGrads(cuTensorBase* X, cuTensorBase* Y,  int rangeH, int rangeW,
                                 int padH, int padW, int strideH, int strideW){
        hipSetDevice(X->data->deviceID);
    
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    rangeH,
                                    rangeW,
                                    padH,
                                    padW,
                                    strideH,
                                    strideW);
    
        float alpha = 1.0f, beta = 1.0f;
    
        checkCUDNN(hipdnnPoolingBackward(
                cudnnHdlG,
                poolDesc,
                &alpha,
                Y->desc.cudnnDesc,
                Y->data->data,
                Y->desc.cudnnDesc,
                Y->grad->data,
                X->desc.cudnnDesc,
                X->data->data,
                &beta,
                X->desc.cudnnDesc,
                X->grad->data
        ))
    
        hipdnnDestroyPoolingDescriptor(poolDesc);
        
        return X;
    }
    
    cuTensorBase *globalAvgPoolOp(cuTensorBase* X, cuTensorBase* Y){
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
    
        hipSetDevice(X->data->deviceID);
    
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    (int)X->desc.sizes.h,
                                    (int)X->desc.sizes.w,
                                    0,
                                    0,
                                    1,
                                    1);
    
        float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnPoolingForward(
                cudnnHdlG,
                poolDesc,
                &alpha,
                X->desc.cudnnDesc,
                X->data->data,
                &beta,
                Y->desc.cudnnDesc,
                Y->data->data
        ))
    
        hipdnnDestroyPoolingDescriptor(poolDesc);
    
        return Y;
    }
    
    cuTensorBase *globalAvgPoolOpGrads(cuTensorBase* X, cuTensorBase* Y){
        hipSetDevice(X->data->deviceID);
    
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    (int)X->desc.sizes.h,
                                    (int)X->desc.sizes.w,
                                    0,
                                    0,
                                    1,
                                    1);
    
        float alpha = 1.0f, beta = 1.0f;
    
        checkCUDNN(hipdnnPoolingBackward(
                cudnnHdlG,
                poolDesc,
                &alpha,
                Y->desc.cudnnDesc,
                Y->data->data,
                Y->desc.cudnnDesc,
                Y->grad->data,
                X->desc.cudnnDesc,
                X->data->data,
                &beta,
                X->desc.cudnnDesc,
                X->grad->data
        ))
    
        hipdnnDestroyPoolingDescriptor(poolDesc);
        
        return X;
    }
} // dylann