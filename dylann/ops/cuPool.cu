//
// Created by Dylan on 9/4/2022.
//

#include "cuPool.cuh"

namespace dylann {
    cuTensorBase *maxPoolOp(cuTensorBase* X, cuTensorBase* Y, int rangeH, int rangeW,
                            int padH, int padW, int strideH, int strideW, float alpha1, float alpha2) {
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
        
        hipSetDevice(X->data->deviceID);
        
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_MAX,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    rangeH,
                                    rangeW,
                                    padH,
                                    padW,
                                    strideH,
                                    strideW);
    
        //float alpha = 1.0f, beta = 0.0f;
        
        checkCUDNN(hipdnnPoolingForward(
                cudnnHdlG,
                poolDesc,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
                ))
    
        checkCUDNN(hipdnnDestroyPoolingDescriptor(poolDesc));
        
        return Y;
    }
    
    cuTensorBase *maxPoolOpGrads(cuTensorBase* X, cuTensorBase* Y,  int rangeH, int rangeW,
                                 int padH, int padW, int strideH, int strideW, float alpha1, float alpha2) {
        hipSetDevice(X->data->deviceID);
    
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_MAX,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    rangeH,
                                    rangeW,
                                    padH,
                                    padW,
                                    strideH,
                                    strideW);
        
        //float alpha = 1.0f, beta = 1.0f;
        
        checkCUDNN(hipdnnPoolingBackward(
                cudnnHdlG,
                poolDesc,
                &alpha1,
                Y->desc.cudnnDesc,
                Y->data->data,
                Y->desc.cudnnDesc,
                Y->grad->data,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->grad->data
                ))
        
        hipdnnDestroyPoolingDescriptor(poolDesc);
        
        return X;
    }
    
    cuTensorBase *avgPoolOp(cuTensorBase* X, cuTensorBase* Y, int rangeH, int rangeW,
                            int padH, int padW, int strideH, int strideW, float alpha1, float alpha2) {
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
    
        hipSetDevice(X->data->deviceID);
    
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    rangeH,
                                    rangeW,
                                    padH,
                                    padW,
                                    strideH,
                                    strideW);
    
        //float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnPoolingForward(
                cudnnHdlG,
                poolDesc,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
        ))
    
        hipdnnDestroyPoolingDescriptor(poolDesc);
    
        return Y;
    }
    
    cuTensorBase *avgPoolOpGrads(cuTensorBase* X, cuTensorBase* Y,  int rangeH, int rangeW,
                                 int padH, int padW, int strideH, int strideW, float alpha1, float alpha2) {
        hipSetDevice(X->data->deviceID);
    
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    rangeH,
                                    rangeW,
                                    padH,
                                    padW,
                                    strideH,
                                    strideW);
    
        //float alpha = 1.0f, beta = 1.0f;
    
        checkCUDNN(hipdnnPoolingBackward(
                cudnnHdlG,
                poolDesc,
                &alpha1,
                Y->desc.cudnnDesc,
                Y->data->data,
                Y->desc.cudnnDesc,
                Y->grad->data,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->grad->data
        ))
    
        hipdnnDestroyPoolingDescriptor(poolDesc);
        
        return X;
    }
    
    cuTensorBase *globalAvgPoolOp(cuTensorBase* X, cuTensorBase* Y, float alpha1, float alpha2) {
        assertAllocated({X, Y});
        assertOnSameDev({X, Y});
    
        hipSetDevice(X->data->deviceID);
    
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    (int)X->desc.sizes.h,
                                    (int)X->desc.sizes.w,
                                    0,
                                    0,
                                    1,
                                    1);
    
        //float alpha = 1.0f, beta = 0.0f;
    
        checkCUDNN(hipdnnPoolingForward(
                cudnnHdlG,
                poolDesc,
                &alpha1,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                Y->desc.cudnnDesc,
                Y->data->data
        ))
    
        hipdnnDestroyPoolingDescriptor(poolDesc);
    
        return Y;
    }
    
    cuTensorBase *globalAvgPoolOpGrads(cuTensorBase* X, cuTensorBase* Y, float alpha1, float alpha2) {
        hipSetDevice(X->data->deviceID);
    
        hipdnnPoolingDescriptor_t poolDesc;
        hipdnnCreatePoolingDescriptor(&poolDesc);
        hipdnnSetPooling2dDescriptor(poolDesc,
                                    HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
                                    HIPDNN_NOT_PROPAGATE_NAN,
                                    (int)X->desc.sizes.h,
                                    (int)X->desc.sizes.w,
                                    0,
                                    0,
                                    1,
                                    1);
    
        //float alpha = 1.0f, beta = 1.0f;
    
        checkCUDNN(hipdnnPoolingBackward(
                cudnnHdlG,
                poolDesc,
                &alpha1,
                Y->desc.cudnnDesc,
                Y->data->data,
                Y->desc.cudnnDesc,
                Y->grad->data,
                X->desc.cudnnDesc,
                X->data->data,
                &alpha2,
                X->desc.cudnnDesc,
                X->grad->data
        ))
    
        hipdnnDestroyPoolingDescriptor(poolDesc);
        
        return X;
    }
} // dylann