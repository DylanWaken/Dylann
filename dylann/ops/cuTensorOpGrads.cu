#include "cuTensorOpGrads.cuh"

namespace dylann{
    cuTensorBase* copy(cuTensorBase* A, cuTensorBase*B){
        assertAllocated({A, B});
        hipSetDevice(A->data->deviceID);
        
        hipMemcpy(B->data->data, A->data->data, A->data->memSize, hipMemcpyDeviceToDevice);
        assertCuda(__FILE__, __LINE__);
        return B;
    }
    
    cuTensorBase* mergeGradBuf(cuTensorBase* A){
        assertAllocated({A});
        assert(A->desc.withGradBuf);
        hipSetDevice(A->data->deviceID);
        
        float a = 1, b = 1;
        checkCUDNN(hipdnnAddTensor(cudnnHdlG,
                                  &b,
                                  A->desc.cudnnDesc,
                                  A->gradBuf->data,
                                  &a,
                                  A->desc.cudnnDesc,
                                  A->grad->data
        ))
        return A;
    }
    
    void GRAD_ADD_A::backwardCalc(cuTensorBase *current) {
        assert(current->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                    current->desc.cudnnDesc,
                                    current->grad->data,
                                    &alpha));
    }
    
    void GRAD_ADD_B::backwardCalc(cuTensorBase *current) {
        assert(target->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        if (target->desc.withGradBuf){
            hipMemcpy(target->gradBuf->data, current->grad->data, target->grad->memSize, hipMemcpyDeviceToDevice);
            assertCuda(__FILE__, __LINE__);
        
            checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                        target->desc.cudnnDesc,
                                        target->gradBuf->data,
                                        &beta))
            mergeGradBuf(target);
            return;
        }
    
        hipMemcpy(target->grad->data, current->grad->data, current->grad->memSize, hipMemcpyDeviceToDevice);
        assertCuda(__FILE__, __LINE__);
    
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                    target->desc.cudnnDesc,
                                    target->grad->data,
                                    &beta))
    }
    
    void GRAD_SCALE::backwardCalc(cuTensorBase *current) {
        assert(current->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                    current->desc.cudnnDesc,
                                    current->grad->data,
                                    &alpha));
    }
}