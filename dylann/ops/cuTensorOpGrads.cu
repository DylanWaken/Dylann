#include "cuTensorOpGrads.cuh"

namespace dylann{
    cuTensorBase* copy(cuTensorBase* A, cuTensorBase*B){
        assertAllocated({A, B});
        hipSetDevice(A->data->deviceID);
        
        hipMemcpy(B->data->data, A->data->data, A->data->memSize, hipMemcpyDeviceToDevice);
        assertCuda(__FILE__, __LINE__);
        return B;
    }
    
    void GRAD_ADD_A::backwardCalc(cuTensorBase *current) {
        assert(current->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                    current->desc.cudnnDesc,
                                    current->grad->data,
                                    &alpha));
    }
    
    void GRAD_ADD_B::backwardCalc(cuTensorBase *current) {
        assert(target->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        float a = 1.0f;
        checkCUDNN(hipdnnAddTensor(cudnnHdlG,
                                  &beta,
                                  current->desc.cudnnDesc,
                                  current->grad->data,
                                  &a,
                                  target->desc.cudnnDesc,
                                  target->grad->data))
    }
    
    void GRAD_SCALE::backwardCalc(cuTensorBase *current) {
        assert(current->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                    current->desc.cudnnDesc,
                                    current->grad->data,
                                    &alpha));
    }
}