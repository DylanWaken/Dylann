#include "cuTensorOpGrads.cuh"

namespace dylann{
    cuTensorBase* copy(cuTensorBase* A, cuTensorBase*B){
        assertAllocated({A, B});
        hipSetDevice(A->data->deviceID);
        
        hipMemcpy(B->data->data, A->data->data, A->data->memSize, hipMemcpyDeviceToDevice);
        assertCuda(__FILE__, __LINE__);
        return B;
    }
    
    cuTensorBase* mergeGradBuf(cuTensorBase* A){
        assertAllocated({A});
        assert(A->desc.withGradBuf);
        hipSetDevice(A->data->deviceID);
        
        float a = 1, b = 1;
        checkCUDNN(hipdnnAddTensor(cudnnHdlG,
                                  &b,
                                  A->desc.cudnnDesc,
                                  A->gradBuf->data,
                                  &a,
                                  A->desc.cudnnDesc,
                                  A->grad->data
        ))
        return A;
    }
    
    void GRAD_ADD_A::backward(cuTensorBase *current) {
        assert(current->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                    current->desc.cudnnDesc,
                                    current->grad->data,
                                    &alpha));
    }
    
    void GRAD_ADD_B::backward(cuTensorBase *current) {
        assert(gradSrc->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        if (gradSrc->desc.withGradBuf){
            hipMemcpy(gradSrc->gradBuf->data, current->grad->data, gradSrc->grad->memSize, hipMemcpyDeviceToDevice);
            assertCuda(__FILE__, __LINE__);
        
            checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                        gradSrc->desc.cudnnDesc,
                                        gradSrc->gradBuf->data,
                                        &beta))
            mergeGradBuf(gradSrc);
            return;
        }
    
        hipMemcpy(gradSrc->grad->data, current->grad->data, current->grad->memSize, hipMemcpyDeviceToDevice);
        assertCuda(__FILE__, __LINE__);
    
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                    gradSrc->desc.cudnnDesc,
                                    gradSrc->grad->data,
                                    &beta))
    }
    
    void GRAD_SCALE::backward(cuTensorBase *current) {
        assert(current->desc.withGrad);
        hipSetDevice(current->data->deviceID);
    
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                                    current->desc.cudnnDesc,
                                    current->grad->data,
                                    &alpha));
    }
}