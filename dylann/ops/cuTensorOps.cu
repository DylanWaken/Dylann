//
// Created by Dylan on 8/5/2022.
//

#include "cuTensorOps.cuh"

namespace dylann{
    
    cuTensorBase* add(cuTensorBase* A, cuTensorBase* B, float alpha, float beta){
        assertAllocated({A, B});
        assertOnSameDev({A, B});
        hipSetDevice(A->data->deviceID);
        
        checkCUDNN(hipdnnAddTensor(cudnnHdlG,
                       &alpha,
                       A->desc.cudnnDesc,
                       A->data->data,
                       &beta,
                       B->desc.cudnnDesc,
                       B->data->data))
        return A;
    }
    
    cuTensorBase* scale(cuTensorBase* A, float alpha){
        assertAllocated({A});
        hipSetDevice(A->data->deviceID);
        
        checkCUDNN(hipdnnScaleTensor(cudnnHdlG,
                       A->desc.cudnnDesc,
                       A->data->data,
                       &alpha))
        return A;
    }
}