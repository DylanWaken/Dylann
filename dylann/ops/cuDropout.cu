//
// Created by Dylan on 9/7/2022.
//

#include "cuDropout.cuh"

namespace dylann {
    cuTensorBase* dropoutOp(cuTensorBase* X, cuTensorBase* Y, float b){
    
        //TODO: created device specific workspace buffer
        assertAllocated({X});
        hipSetDevice(X->data->deviceID);
        
        hipdnnDropoutDescriptor_t dropoutDesc;
        hipdnnCreateDropoutDescriptor(&dropoutDesc);
        checkCUDNN(hipdnnSetDropoutDescriptor(
                dropoutDesc,
                cudnnHdlG,
                b,
                nullptr,
                0,
                time(nullptr)
                ))
    
        checkCUDNN(cudnnDropoutForward(
                    cudnnHdlG,
                    dropoutDesc,
                    X->desc.cudnnDesc,
                    X->data->data,
                    Y->desc.cudnnDesc,
                    Y->data->data,
                    cudnnWorkspaceG,
                CUDNN_WORKSPACE_SIZE_G
                ))
            
        hipdnnDestroyDropoutDescriptor(dropoutDesc);
        return Y;
    }
    
    cuTensorBase* dropoutOpGrads(cuTensorBase* X, cuTensorBase* Y, float b){
    
        hipSetDevice(X->data->deviceID);
        
        hipdnnDropoutDescriptor_t dropoutDesc;
        hipdnnCreateDropoutDescriptor(&dropoutDesc);
        checkCUDNN(hipdnnSetDropoutDescriptor(
                dropoutDesc,
                cudnnHdlG,
                b,
                nullptr,
                0,
                time(nullptr)
                ))
    
        checkCUDNN(cudnnDropoutBackward(
                    cudnnHdlG,
                    dropoutDesc,
                    Y->desc.cudnnDesc,
                    Y->grad->data,
                    X->desc.cudnnDesc,
                    X->grad->data,
                    cudnnWorkspaceG,
                CUDNN_WORKSPACE_SIZE_G
                ))
            
        hipdnnDestroyDropoutDescriptor(dropoutDesc);
        return X;
    }
} // dylann