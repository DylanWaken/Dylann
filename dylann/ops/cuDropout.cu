//
// Created by Dylan on 9/7/2022.
//

#include "cuDropout.cuh"
#include <chrono>

namespace dylann {
    cuTensorBase* dropoutOp(cuTensorBase* X, cuTensorBase* Y, cuTensorBase* mask, float b){
    
        //TODO: created device specific workspace buffer
        assertAllocated({X});
        hipSetDevice(X->data->deviceID);
        
        hipdnnDropoutDescriptor_t dropoutDesc;
        hipdnnCreateDropoutDescriptor(&dropoutDesc);
        checkCUDNN(hipdnnSetDropoutDescriptor(
                dropoutDesc,
                cudnnHdlG,
                b,
                nullptr,
                0,
                chrono::system_clock::now().time_since_epoch().count()
                ))
    
        checkCUDNN(cudnnDropoutForward(
                    cudnnHdlG,
                    dropoutDesc,
                    X->desc.cudnnDesc,
                    X->data->data,
                    Y->desc.cudnnDesc,
                    Y->data->data,
                    mask->data->data,
                mask->data->memSize
                ))
            
        hipdnnDestroyDropoutDescriptor(dropoutDesc);
        return Y;
    }
    
    cuTensorBase* dropoutOpGrads(cuTensorBase* X, cuTensorBase* Y, cuTensorBase* reserved, float b){
    
        hipSetDevice(X->data->deviceID);
        
        hipdnnDropoutDescriptor_t dropoutDesc;
        hipdnnCreateDropoutDescriptor(&dropoutDesc);
        checkCUDNN(hipdnnSetDropoutDescriptor(
                dropoutDesc,
                cudnnHdlG,
                b,
                nullptr,
                0,
                time(nullptr)
                ))
    
        checkCUDNN(cudnnDropoutBackward(
                    cudnnHdlG,
                    dropoutDesc,
                    Y->desc.cudnnDesc,
                    Y->grad->data,
                    X->desc.cudnnDesc,
                    X->grad->data,
                    cudnnWorkspaceG,
                CUDNN_WORKSPACE_SIZE_G
                ))
            
        hipdnnDestroyDropoutDescriptor(dropoutDesc);
        return X;
    }
} // dylann