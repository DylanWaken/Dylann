//
// Created by Dylan on 9/23/2022.
//

#include "Data.cuh"

HostTensorBase *HostTensorBase::create(dylann::shape4 sizes, hipdnnDataType_t dataType) {
    HostTensorBase* tensor;
    hipHostMalloc(&tensor, sizeof(HostTensorBase));
    assertCuda(__FILE__, __LINE__);
    
    tensor->sizes = sizes;
    tensor->dataType = dataType;
    tensor->elemSize = dylann::sizeOfDtype(dataType);
    
    hipHostMalloc(&tensor->data, tensor->sizes.size * tensor->elemSize);
    assertCuda(__FILE__, __LINE__);
    
    return tensor;
}
