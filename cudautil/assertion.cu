//
// Created by Dylan on 8/5/2022.
//

#include "assertion.cuh"

void assertCuda(const char *file, int line){
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        logFatal(io::LOG_SEG_COMP, "CUDA error at " + string(file) + ": " + to_string(line)
        + ": " + hipGetErrorString(err));
        std::cerr << "CUDA error at " << file << ":" << line << ": " << hipGetErrorString(err) << std::endl;
        assert(false);
    }
}