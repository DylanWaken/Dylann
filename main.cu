#include <iostream>

#include "presets/Resnet.cuh"
#include "dylann/module/Sequence.cuh"

#include <thread>
#include <mutex>
#include <condition_variable>
#include "io/dataset/Dataset.cuh"
#include "presets/readFuncs/BuildinReadfuncs.cuh"
#define MINI_BATCH_SIZE 64

using namespace dylann;
using namespace std;
using namespace io;

int main() {


    //register model
    initEngineContext();
    auto X0 = cuTensor::create(0, HIPDNN_DATA_FLOAT, {MINI_BATCH_SIZE, 3, 32, 32});
    
    ResnetIdentity id = ResnetIdentity();
    ResnetConv cv = ResnetConv();
    
    auto X = conv2D(X0, 3, 3, 64, 1, 1, 1, 1, 1, 1);
    X = batchnorm2d(X, 1e-8, 1);
    X = relu(X);

    for(auto i = 0; i < 5; i++) X = id.forward(X);
    X = cv.forward(X);

    for(auto i = 0; i < 5; i++) X = id.forward(X);
    X = cv.forward(X);

    for(auto i = 0; i < 6; i++) X = id.forward(X);

    auto X2 = flatten(X);
    X2 = linear(X2, 1024);
    X2 = relu(X2);
    X2 = linear(X2, 256);
    X2 = relu(X2);
    auto X3 = linear(X2, 10);
    auto Y = softmaxCE(X3, 10);

    auto seq = ctx2seq();
    seq->generateGrad();
    seq->setLoss(new CrossEntropy(Y.impl));
    seq->setOpt(new Momentum(0.01/64));
    seq->randomizeParams();

    for(auto& i : seq->forwardOpSeq){
        i->print();
    }

    for(auto& i : seq->backwardOpSeq){
        i->print();
    }


    auto* dataset = new DatasetCV(50000, 6400, MINI_BATCH_SIZE, 16,6400,
                                       {1, 3, 32, 32},
                                       shape4(10), HIPDNN_DATA_FLOAT);

    ReadFuncCV* readFunc = new CIFAR_10ReadFunc(R"(D:\Resources\Datasets\cifar-10-bin)", 16);
    dataset->bindReadFunc(readFunc);
    dataset->bindAugCV({
        new RandFlip(),
        new RandPadCorp(4),
    });
    dataset->bindAugTensor({new UniformNorm(0, 1)});
    dataset->construct();

    auto label = cuTensor::create(0, HIPDNN_DATA_FLOAT, {MINI_BATCH_SIZE, 1, 1,10});
    
    float runningLoss = 0;
    for(int i = 0; i < 500000; i++){
        dataset->nextMiniBatch(X0.impl, label.impl);
        seq->forward();
        
        float loss = seq->getLoss(label.impl);
        runningLoss += loss;
        seq->backward(label.impl);
        seq->opt->apply();
        
        seq->resetGrad();
        
        if(i % 100 == 0 && i != 0){
            cout << runningLoss / 100 << ", ";
            runningLoss = 0;
            
            float valLoss = 0;
            for(int j = 0; j < 50; j++){
                dataset->nextValBatch(X0.impl, label.impl);
                seq->forward();
                valLoss += seq->getLoss(label.impl);
                seq->resetGrad();
            }
            cout << valLoss / 50 << ", ";
        }
    }
}
